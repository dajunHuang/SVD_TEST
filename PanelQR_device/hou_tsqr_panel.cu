#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "TallShinnyQR.h"

#define NUM_WARPUP 2
#define NUM_REPEAT 5

template <typename T>
void test_hou_tsqr_panel(size_t m, size_t n) {
    // hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const size_t lda = m;
    const size_t ldr = n;
    const size_t ldy = m;

    std::vector<T> A(m * n, 0);
    std::vector<T> A_from_gpu(m * n, 0);
    std::vector<T> Y_from_gpu(m * n, 0);
    std::vector<T> R_from_gpu(n * n, 0);

    std::default_random_engine eng(0U);
    // std::uniform_int_distribution<int> dis(0, 5);
    std::uniform_real_distribution<T> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    std::generate(A.begin(), A.end(), rand);

    T *d_A = nullptr;
    T *d_Y = nullptr;
    T *d_R = nullptr;
    T *d_work = nullptr;

    /* step 1: create cusolver handle, bind a stream */
    // CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(T) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Y), sizeof(T) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(T) * n * n));

    const unsigned int blockNum = (m + 128U - 1U) / 128U;
    const size_t ldwork{32U * blockNum};

    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(T) * ldwork * 32));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(T) * A.size(),
                               hipMemcpyHostToDevice, stream));

    CUDA_CHECK(hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice));
    hou_tsqr_panel<T, 128, 32>(cublasH, m, n, d_A, lda, d_Y, ldy, d_R, ldr,
                               d_work, ldwork);
    CUDA_CHECK_LAST_ERROR();

    // hipEvent_t start, stop;
    // float time = 0, temp_time = 0;

    // CUDA_CHECK(hipEventCreate(&start));
    // CUDA_CHECK(hipEventCreate(&stop));
    // for(int i{0}; i < NUM_WARPUP; ++i)
    // {
    //     hipMemcpy(d_A, A.data(), sizeof(T) * A.size(),
    //     hipMemcpyHostToDevice); hou_tsqr_panel<T, 128, 32>(cublasH, m, n,
    //     d_A, lda, d_R, ldr, d_work);
    // }
    // CUDA_CHECK(hipStreamSynchronize(stream));
    // for(int i{0}; i < NUM_REPEAT; ++i)
    // {
    //     hipMemcpy(d_A, A.data(), sizeof(T) * A.size(),
    //     hipMemcpyHostToDevice); CUDA_CHECK(hipEventRecord(start, stream));

    //     hou_tsqr_panel<T, 128, 32>(cublasH, m, n, d_A, lda, d_R, ldr,
    //     d_work);

    //     CUDA_CHECK(hipEventRecord(stop, stream));
    //     CUDA_CHECK(hipEventSynchronize(stop));
    //     CUDA_CHECK_LAST_ERROR();
    //     CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    //     time += temp_time;
    // }
    // time /= NUM_REPEAT;

    CUDA_CHECK(hipMemcpyAsync(A_from_gpu.data(), d_A,
                               sizeof(T) * A_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(Y_from_gpu.data(), d_Y,
                               sizeof(T) * Y_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(R_from_gpu.data(), d_R,
                               sizeof(T) * R_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    // std::cout << "hou_tsqr_panel Latency: " << time << " ms" << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_Y));
    CUDA_CHECK(hipFree(d_R));
    CUDA_CHECK(hipFree(d_work));

    // CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
}

template void test_hou_tsqr_panel<float>(size_t m, size_t n);
template void test_hou_tsqr_panel<double>(size_t m, size_t n);

int main(int argc, char *argv[]) {
    size_t m = 2048, n = 32;
    int dataType = 1;

    print_device_info();

    if (argc >= 4) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        dataType = atoi(argv[3]);
    }

    if (0 == dataType) {
        // test_hou_tsqr_panel<half>(m, n);
    } else if (1 == dataType) {
        test_hou_tsqr_panel<float>(m, n);
    } else if (2 == dataType) {
        // test_hou_tsqr_panel<double>(m, n);
    }

    return 0;
}
