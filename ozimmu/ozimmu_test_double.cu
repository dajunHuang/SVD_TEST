/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

#include <algorithm>
#include <iostream>
#include "tc_syr2k.h"
#include "tc_ozimmu_syr2k.h"

#define NUM_WARPUP 2
#define NUM_REPEAT 5

template <typename T>
void random_initialize_matrix(T* A, size_t m, size_t n, size_t lda,
                              unsigned int seed = 0U)
{
    std::default_random_engine eng(seed);
    // The best way to verify is to use integer values.
    std::uniform_int_distribution<int> dis(0, 5);
    // std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    for (size_t j{0U}; j < n; ++j)
    {
        for (size_t i{0U}; i < m; ++i)
        {
            A[i + j * lda] = static_cast<T>(rand());
        }
    }
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH;
    hipStream_t stream;

    int n = 8192, k = 128, nb = 128;

    if(argc >= 4)
    {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
    }

    const int lda = n;
    const int ldb = n;
    const int ldc = n;

    std::vector<double> A(n * k, 0);
    std::vector<double> B(n * k, 0);
    std::vector<double> C(n * n, 0);

    std::default_random_engine eng(0U);
    // std::uniform_int_distribution<int> dis(0, 5);
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    std::generate(A.begin(), A.end(), rand);
    std::generate(B.begin(), B.end(), rand);

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_C = nullptr;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * C.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    float time;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for(int i{0}; i < NUM_WARPUP; ++i)
    {
        tc_ozimmu_syr2k(cublasH, n, k, 1.0, d_A, lda, d_B, ldb, 0.0, d_C, ldc, nb);
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipEventRecord(start, stream));
    for(int i{0}; i < NUM_REPEAT; ++i)
    {
        tc_ozimmu_syr2k(cublasH, n, k, 1.0, d_A, lda, d_B, ldb, 0.0, d_C, ldc, nb);
    }
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));
    time /= NUM_REPEAT;

    CUDA_CHECK(
        hipMemcpyAsync(C.data(), d_C, sizeof(double) * C.size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    // std::printf("after gesvd: info_gpu = %d\n", info_gpu);
    // if (0 == info_gpu) {
    //     std::printf("gesvd converges \n");
    // } else if (0 > info_gpu) {
    //     std::printf("%d-th parameter is wrong \n", -info_gpu);
    //     exit(1);
    // } else {
    //     std::printf("WARNING: info = %d : gesvd does not converge \n", info_gpu);
    // }

    // std::printf("S = singular values (matlab base-1)\n");
    // print_matrix(n, 1, S.data(), n);
    // std::printf("=====\n");

    // std::printf("U = left singular vectors (matlab base-1)\n");
    // print_matrix(m, m, U.data(), ldu);
    // std::printf("=====\n");

    // std::printf("VT = right singular vectors (matlab base-1)\n");
    // print_matrix(n, n, VT.data(), ldvt);
    // std::printf("=====\n");

    std::cout << "TCGEMMS (Double) Latency: " << time << " ms" << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
