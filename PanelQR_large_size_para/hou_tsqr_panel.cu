#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "TallShinnyQR.h"
#include "TallShinnyQR_ori.h"

#define NUM_WARPUP 20
#define NUM_REPEAT 50

template <typename T>
void test_tsqr(int block_size, int m, int n) {
    // hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int lda = m;
    const int ldr = n;
    double kOne = 1, kZero = 0;

    std::vector<T> A(m * n, 0);
    std::vector<T> A_from_gpu(m * n, 0);
    std::vector<T> R_from_gpu(n * n, 0);

    std::default_random_engine eng(0U);
    // std::uniform_int_distribution<int> dis(0, 5);
    std::uniform_real_distribution<T> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    std::generate(A.begin(), A.end(), rand);

    T *d_A = nullptr;
    T *d_R = nullptr;
    T *d_work1 = nullptr, *d_work2 = nullptr, *d_work_ori = nullptr;

    /* step 1: create cusolver handle, bind a stream */
    // CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(T) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(T) * n * n));
    // CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work_ori), sizeof(T) *
    // m * m));

    int ldwork1 = m, ldwork2;
    int pre_reduction_time =
        ceil((log(m) - log(13824)) / (log(block_size) - log(n)));
    if (pre_reduction_time > 0) {
        ldwork2 = 13824;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work1),
                              pre_reduction_time * sizeof(T) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work2),
                              sizeof(T) * 13824 * n));
    } else {
        ldwork2 = m;
        CUDA_CHECK(
            hipMalloc(reinterpret_cast<void **>(&d_work2), sizeof(T) * m * n));
    }

    // # origianl
    // CUDA_CHECK(hipMemcpy(d_A, A.data(), sizeof(T) * A.size(),
    //                       hipMemcpyHostToDevice));
    // printf("\nhou_tsqr_panel_ori\n");
    // hou_tsqr_panel_ori<T, 128, 32>(cublasH, m, n, d_A, lda, d_R, ldr,
    //                                d_work_ori);
    // CUDA_CHECK(hipDeviceSynchronize());
    // CUDA_CHECK_LAST_ERROR();
    // printf("R\n");
    // printDeviceMatrixV2(d_R, ldr, 32, 32);
    // printf("Q\n");
    // printDeviceMatrixV2(d_A, lda, m < 169 ? m : 169, 32);

    CUDA_CHECK(hipMemcpy(d_A, A.data(), sizeof(T) * A.size(),
                          hipMemcpyHostToDevice));
    // printf("A\n");
    // printDeviceMatrixV2(d_A, lda, 1024, n);
    // printf("tsqr\n");
    tsqr<T>(cublasH, block_size, m, n, d_A, lda, d_R, ldr, d_work1, ldwork1,
            d_work2, ldwork2);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK_LAST_ERROR();
    // printf("R\n");
    // printDeviceMatrixV2(d_R, ldr, 32, 32);
    // printf("Q\n");
    // printDeviceMatrixV2(d_A, lda, m < 32 ? m : 32, n < 32 ? n : 32);

    T *d_Q = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q), sizeof(T) * n * n));
    hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &kOne, d_A, lda,
                d_A, lda, &kZero, d_Q, n);
    // printDeviceMatrixV2(d_Q, n, n, n);

    hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, n, &kOne, d_A, lda,
                d_R, ldr, &kZero, d_A, lda);
    CUDA_CHECK(hipDeviceSynchronize());
    // printf("Q * R\n");
    // printDeviceMatrixV2(d_A, lda, 1024, n);

    CUDA_CHECK(hipMemcpyAsync(A_from_gpu.data(), d_A,
                               sizeof(T) * A_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    if (!all_close(A_from_gpu.data(), A.data(), m, n, lda, 1.0e-4, 1.0e-5)) {
        std::cout << "Error: tsqr" << std::endl;
        exit(-1);
    }

    hipEvent_t start, stop;
    float time = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice);
        CUDA_CHECK(hipDeviceSynchronize());
        // printf("warmup %d\n", i);
        tsqr<T>(cublasH, block_size, m, n, d_A, lda, d_R, ldr, d_work1, ldwork1,
                d_work2, ldwork2);
        CUDA_CHECK(hipDeviceSynchronize());
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice);
        CUDA_CHECK(hipDeviceSynchronize());
        // printf("repeat %d\n", i);
        CUDA_CHECK(hipEventRecord(start, stream));

        tsqr<T>(cublasH, block_size, m, n, d_A, lda, d_R, ldr, d_work1, ldwork1,
                d_work2, ldwork2);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time += temp_time;
    }
    time /= NUM_REPEAT;

    CUDA_CHECK(hipMemcpyAsync(A_from_gpu.data(), d_A,
                               sizeof(T) * A_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(R_from_gpu.data(), d_R,
                               sizeof(T) * R_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "tsqr Latency: " << time << " ms" << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_R));
    CUDA_CHECK(hipFree(d_work_ori));
    CUDA_CHECK(hipFree(d_work1));
    CUDA_CHECK(hipFree(d_work2));

    // CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
}

// template void test_tsqr<float>(int m, int n);
template void test_tsqr<double>(int block_size, int m, int n);

int main(int argc, char *argv[]) {
    int m = 13824, n = 32;
    int block_size = 128;
    int dataType = 2;

    // print_device_info();

    if (argc >= 4) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        dataType = atoi(argv[3]);
    }

    if (0 == dataType) {
        // test_tsqr<half>(m, n);
    } else if (1 == dataType) {
        // test_tsqr<float>(m, n);
    } else if (2 == dataType) {
        test_tsqr<double>(block_size, m, n);
    }

    return 0;
}
