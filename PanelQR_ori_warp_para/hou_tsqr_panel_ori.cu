#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "TallShinnyQR_ori.h"

#define NUM_WARPUP 2
#define NUM_REPEAT 5

template <typename T>
void test_tsqr(long m, long n) {
    // hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const long lda = m;
    const long ldr = n;

    std::vector<T> A(m * n, 0);
    std::vector<T> A_from_gpu(m * n, 0);
    std::vector<T> R_from_gpu(n * n, 0);

    std::default_random_engine eng(0U);
    // std::uniform_int_distribution<int> dis(0, 5);
    std::uniform_real_distribution<T> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    std::generate(A.begin(), A.end(), rand);

    T *d_A = nullptr;
    T *d_R = nullptr;
    T *d_work = nullptr;

    /* step 1: create cusolver handle, bind a stream */
    // CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(T) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(T) * n * n));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(T) * m * m));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(T) * A.size(),
                               hipMemcpyHostToDevice, stream));

    hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice);
    tsqr<T>(cublasH, m, n, d_A, lda, d_R, ldr, d_work);
    CUDA_CHECK_LAST_ERROR();

    check_QR_accuracy<T>(m, n, d_A, lda, d_R, ldr, A);

    hipEvent_t start, stop;
    float time = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice);
        CUDA_CHECK(hipDeviceSynchronize());
        tsqr<T>(cublasH, m, n, d_A, lda, d_R, ldr, d_work);
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        hipMemcpy(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipEventRecord(start, stream));

        tsqr<T>(cublasH, m, n, d_A, lda, d_R, ldr, d_work);

        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time += temp_time;
    }
    time /= NUM_REPEAT;

    std::cout << "hou_tsqr_panel Latency: " << time << " ms" << std::endl;

    CUDA_CHECK(hipMemcpyAsync(A_from_gpu.data(), d_A,
                               sizeof(T) * A_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(R_from_gpu.data(), d_R,
                               sizeof(T) * R_from_gpu.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_R));
    CUDA_CHECK(hipFree(d_work));

    // CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
}

template void test_tsqr<float>(long m, long n);
template void test_tsqr<double>(long m, long n);

int main(int argc, char *argv[]) {
    long m = 13824, n = 32, dataType = 2;

    if (argc >= 4) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        dataType = atoi(argv[3]);
    }

    if (0 == dataType) {
        // test_hou_tsqr_panel<half>(m, n);
    } else if (1 == dataType) {
        test_tsqr<float>(m, n);
    } else if (2 == dataType) {
        test_tsqr<double>(m, n);
    }

    return 0;
}
