/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

#include <algorithm>
#include <iostream>

#define NUM_WARPUP 2
#define NUM_REPEAT 5

template <typename T>
void random_initialize_matrix(T* A, size_t m, size_t n, size_t lda,
                              unsigned int seed = 0U)
{
    std::default_random_engine eng(seed);
    // The best way to verify is to use integer values.
    std::uniform_int_distribution<int> dis(0, 5);
    // std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    for (size_t j{0U}; j < n; ++j)
    {
        for (size_t i{0U}; i < m; ++i)
        {
            A[i + j * lda] = static_cast<T>(rand());
        }
    }
}

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    int m = 0, n = 0;

    if(argc < 3)
    {
        m = 1024;
        n = 1024;
    }
    else
    {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
    }

    const int lda = m;  // lda >= m
    const int ldu = m;  // ldu >= m
    const int ldvt = n; // ldvt >= n if jobu = 'A'

    std::vector<double> A(m * n, 0);

    std::default_random_engine eng(0U);
    // std::uniform_int_distribution<int> dis(0, 5);
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    auto const rand = [&dis, &eng]() { return dis(eng); };
    std::generate(A.begin(), A.end(), rand);

    std::vector<double> U(ldu * m, 0);  /* m-by-m unitary matrix, left singular vectors  */
    std::vector<double> VT(ldvt * n, 0); /* n-by-n unitary matrix, right singular vectors */
    std::vector<double> S(n, 0);        /* numerical singular value */
    int info_gpu = 0;                                  /* host copy of error info */

    double *d_A = nullptr;
    double *d_S = nullptr;  /* singular values */
    double *d_U = nullptr;  /* left singular vectors */
    double *d_VT = nullptr; /* right singular vectors */

    int *devInfo = nullptr;

    int lwork = 0; /* size of workspace */
    double *d_work = nullptr;
    double *d_rwork = nullptr;

    // std::printf("A = (matlab base-1)\n");
    // print_matrix(m, n, A.data(), lda);
    // std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(double) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VT), sizeof(double) * VT.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&devInfo), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    /* step 3: query working space of SVD */
    CUSOLVER_CHECK(hipsolverDnDgesvd_bufferSize(cusolverH, m, n, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: compute SVD */
    signed char jobu = 'A';  // all m columns of U
    signed char jobvt = 'A'; // all n rows of VT

    hipEvent_t start, stop;
    float time;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for(int i{0}; i < NUM_WARPUP; ++i)
    {
        CUSOLVER_CHECK(hipsolverDnDgesvd(cusolverH, jobu, jobvt, m, n, d_A, lda,
                                        d_S, d_U, ldu, d_VT, ldvt,
                                        d_work, lwork, d_rwork, devInfo));
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipEventRecord(start, stream));
    for(int i{0}; i < NUM_REPEAT; ++i)
    {
        CUSOLVER_CHECK(hipsolverDnDgesvd(cusolverH, jobu, jobvt, m, n, d_A, lda,
                                        d_S, d_U, ldu, d_VT, ldvt,
                                        d_work, lwork, d_rwork, devInfo));
    }
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));
    time /= NUM_REPEAT;

    CUDA_CHECK(
        hipMemcpyAsync(U.data(), d_U, sizeof(double) * U.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(VT.data(), d_VT, sizeof(double) * VT.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(
        hipMemcpyAsync(S.data(), d_S, sizeof(double) * S.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    // std::printf("after gesvd: info_gpu = %d\n", info_gpu);
    // if (0 == info_gpu) {
    //     std::printf("gesvd converges \n");
    // } else if (0 > info_gpu) {
    //     std::printf("%d-th parameter is wrong \n", -info_gpu);
    //     exit(1);
    // } else {
    //     std::printf("WARNING: info = %d : gesvd does not converge \n", info_gpu);
    // }

    // std::printf("S = singular values (matlab base-1)\n");
    // print_matrix(n, 1, S.data(), n);
    // std::printf("=====\n");

    // std::printf("U = left singular vectors (matlab base-1)\n");
    // print_matrix(m, m, U.data(), ldu);
    // std::printf("=====\n");

    // std::printf("VT = right singular vectors (matlab base-1)\n");
    // print_matrix(n, n, VT.data(), ldvt);
    // std::printf("=====\n");

    std::cout << "Cusolver SVD (Double) Latency: " << time << " ms" << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_VT));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(devInfo));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_rwork));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
