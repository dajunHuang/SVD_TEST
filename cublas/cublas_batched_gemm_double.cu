#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_REPEAT 1

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    int n = 32768, k = 32768, nb = 512;

    if (argc >= 3) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
    }

    double *d_A = nullptr;
    double *d_C = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    int lda = n, ldc = n;

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, k);

    hipEvent_t start, stop;
    float time0 = 0, time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start, stream));

        for (int j{0}; j < n / nb; ++j) {
            hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &one,
                         d_A + j * nb, lda, d_A + j * nb,
                         lda, &zero, d_C + j * nb + j * nb * ldc,
                         ldc);
        }

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time0 += temp_time;
    }
    time0 /= NUM_REPEAT;

    std::cout << "[cublas        gemm] " << "n: " << n << ", k: " << k
              << ", "
              << "latency: " << time0 << " ms, "
              << (long)n / nb * 2 * nb * nb * k / time0 / 1e9
              << " TFLOPS" << std::endl;

    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start, stream));

        hipblasDgemmStridedBatched(
            cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &one, d_A,
            lda, nb, d_A, lda, nb, &zero, d_C, ldc,
            nb + nb * ldc, n / nb);

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "[cublas bached gemm] " << "n: " << n << ", k: " << k
              << ", "
              << "latency: " << time1 << " ms, "
              << (long)n / nb * 2 * nb * nb * k / time1 / 1e9
              << " TFLOPS" << std::endl;


    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
