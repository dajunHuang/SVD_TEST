#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>

#include "utils.h"

void generateUniqueRandomArray(int* arr, int nb, int max) {
    for (int i = 0; i < nb; ++i) {
        arr[i] = rand() % max; // 生成 0 到 max-1 之间的随机数
    }
}

template <typename T>
__global__ void swap_kernel(int n, T *x, int incx, T *y, int incy) {}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    int m = 16384, n = 16384, nb = 1024;

    if (argc >= 4) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        nb = atoi(argv[3]);
    }

    double *d_A_ori = nullptr;
    double *d_A_cublas = nullptr;
    double *d_A_custom = nullptr;
    int *hdevIpiv = (int *)malloc(sizeof(int) * nb);

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    int lda = m;

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A_ori), sizeof(double) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A_cublas), sizeof(double) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A_custom), sizeof(double) * lda * n));

    generateUniformMatrixDouble(d_A_ori, lda, n);
    generateUniqueRandomArray(hdevIpiv, nb, m);
    // for(int i = 0; i < nb; ++i) {
    //     printf("%d: %d\n", i, hdevIpiv[i]);
    // }


    // printf("d_A_ori:\n");
    // print_device_matrix(d_A_ori, lda, 16, 16);

    CUDA_CHECK(hipMemcpy(d_A_cublas, d_A_ori, lda * n * sizeof(double), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_A_custom, d_A_ori, lda * n * sizeof(double), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());

    // CUBLAS_CHECK(hipblasDswap(cublasH, n, d_A_cublas + r1, lda, d_A_cublas + r2, lda));

    // printf("d_A_cublas:\n");
    // print_device_matrix(d_A_cublas, lda, 16, 16);

    hipEvent_t start, stop;
    float time = 0;


    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipStreamSynchronize(stream));
    hipDeviceSynchronize();
    CUDA_CHECK(hipEventRecord(start, stream));
    for(int i = 0; i < nb; ++i) {
        hipblasDswap(cublasH, n, d_A_cublas + i, lda, d_A_cublas + hdevIpiv[i], lda);
    }
    hipDeviceSynchronize();
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));
    CUDA_CHECK(hipStreamSynchronize(stream));
    std::cout << "[cublas dswap] " << "m: " << m << ", n: " << n << ", " << "nb: " << nb << ", "
              << "latency: " << time << " ms" << std::endl;


    /* free resources */
    CUDA_CHECK(hipFree(d_A_ori));
    CUDA_CHECK(hipFree(d_A_cublas));
    CUDA_CHECK(hipFree(d_A_custom));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
