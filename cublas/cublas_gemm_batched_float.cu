#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <thrust/device_ptr.h>
#include <thrust/equal.h>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 10

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    int m = 32768, n = 32768, k = 32768;
    int nm = 1024, nn = 1024;

    if (argc >= 6) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        nm = atoi(argv[4]);
        nn = atoi(argv[5]);
    }

    assert(m % nm == 0 && n % nn == 0);

    float *d_A = nullptr;
    float *d_B = nullptr;
    // float *d_C1 = nullptr;
    float *d_C2 = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    int lda = m, ldb = k, ldc = m;

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * ldb * n));
    // CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C1), sizeof(float) * ldc * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C2), sizeof(float) * ldc * n));

    generateUniformMatrix(d_A, lda, k);
    generateUniformMatrix(d_B, ldb, n);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // for (int i{0}; i < NUM_WARPUP; ++i) {
    //     CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, d_A, lda, d_B,
    //                              ldb, &zero, d_C1,
    //                              ldc));  // CUBLAS_GEMM_ALGO0_TENSOR_OP
    // }
    // CUDA_CHECK(hipStreamSynchronize(stream));
    // for (int i{0}; i < NUM_REPEAT; ++i) {
    //     CUDA_CHECK(hipStreamSynchronize(stream));
    //     CUDA_CHECK(hipEventRecord(start, stream));

    //     CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, d_A, lda, d_B,
    //                              ldb, &zero, d_C1,
    //                              ldc));  // CUBLAS_GEMM_ALGO0_TENSOR_OP

    //     CUDA_CHECK(hipStreamSynchronize(stream));
    //     CUDA_CHECK(hipEventRecord(stop, stream));
    //     CUDA_CHECK(hipEventSynchronize(stop));
    //     CUDA_CHECK_LAST_ERROR();
    //     CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    //     time += temp_time;
    // }
    // time /= NUM_REPEAT;

    // CUDA_CHECK(hipStreamSynchronize(stream));

    // std::cout << "[cublas sgemm] " << "m: " << m << ", n: " << n << ", k: " << k << ", "
    //           << "latency: " << time << " ms, "
    //           << "Effective TFLOPS: " << 2.0 * m * n * k / time / 1e9 << " TFLOPS, " << std::endl;

    int loop_m = (m + nm - 1) / nm;
    int loop_n = (n + nn - 1) / nn;
    int loop_num = loop_m * loop_n;

    std::vector<float *> h_Aarray(loop_num);
    std::vector<float *> h_Barray(loop_num);
    std::vector<float *> h_Carray(loop_num);
    for (int i = 0; i < loop_m; ++i) {
        for (int j = 0; j < loop_n; ++j) {
            h_Aarray[i * loop_n + j] = d_A + i * nm;
            h_Barray[i * loop_n + j] = d_B + j * nn * ldc;
            h_Carray[i * loop_n + j] = d_C2 + i * nm + j * nn * ldc;
        }
    }

    float **d_Aarray, **d_Barray, **d_Carray;
    CUDA_CHECK(hipMalloc(&d_Aarray, sizeof(float *) * loop_num));
    CUDA_CHECK(hipMalloc(&d_Barray, sizeof(float *) * loop_num));
    CUDA_CHECK(hipMalloc(&d_Carray, sizeof(float *) * loop_num));
    CUDA_CHECK(
        hipMemcpy(d_Aarray, h_Aarray.data(), sizeof(float *) * loop_num, hipMemcpyHostToDevice));
    CUDA_CHECK(
        hipMemcpy(d_Barray, h_Barray.data(), sizeof(float *) * loop_num, hipMemcpyHostToDevice));
    CUDA_CHECK(
        hipMemcpy(d_Carray, h_Carray.data(), sizeof(float *) * loop_num, hipMemcpyHostToDevice));

    time = 0, temp_time = 0;
    for (int i{0}; i < NUM_WARPUP; ++i) {
        CUBLAS_CHECK(hipblasSgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, nm, nn, k, &one,
                                        d_Aarray, lda, d_Barray, ldb, &zero, d_Carray, ldc,
                                        loop_num));  // CUBLAS_GEMM_ALGO0_TENSOR_OP
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(start, stream));

        CUBLAS_CHECK(hipblasSgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, nm, nn, k, &one,
                                        d_Aarray, lda, d_Barray, ldb, &zero, d_Carray, ldc,
                                        loop_num));  // CUBLAS_GEMM_ALGO0_TENSOR_OP

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time += temp_time;
    }
    time /= NUM_REPEAT;

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "[cublas batched sgemm] " << "m: " << m << ", n: " << n << ", k: " << k << ", "
              << "nm: " << nm << ", nn: " << nn << ", "
              << "latency: " << time << " ms, "
              << "Effective TFLOPS: " << 2.0 * m * n * k / time / 1e9 << " TFLOPS, " << std::endl;

    // thrust::device_ptr<float> thrust_d_C1(d_C1);
    // thrust::device_ptr<float> thrust_d_C2(d_C2);
    // bool are_equal = thrust::equal(thrust_d_C1, thrust_d_C1 + loop_num, thrust_d_C2);
    // if (are_equal) {
    //     std::cout << "C1 C2 equal" << std::endl;
    // } else {
    //     std::cout << "C1 C2 not equal" << std::endl;
    // }

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    // CUDA_CHECK(hipFree(d_C1));
    CUDA_CHECK(hipFree(d_C2));
    CUDA_CHECK(hipFree(d_Aarray));
    CUDA_CHECK(hipFree(d_Barray));
    CUDA_CHECK(hipFree(d_Carray));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
